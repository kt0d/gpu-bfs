#include "hip/hip_runtime.h"
#include "bfs.cuh"

#include "bfs_kernels.cuh"
#include "csr_matrix.h"

#include <algorithm>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

int div_up(int dividend, int divisor)
{
	return (dividend % divisor == 0)?(dividend/divisor):(dividend/divisor+1);
}

__global__ void init_dist_kernel(const int n, int*const distance,const int start)
{
	// Calculate corresponding vertex.
	const int id = blockIdx.x*blockDim.x + threadIdx.x;

	// Fill distance vector with infinity.
	if(id < n)
		distance[id]=bfs::infinity;
	// Set distance to starting vertex to 0.
	if(id == start)
		distance[id]=0;
}

__global__ void init_bitmask(const int count, hipSurfaceObject_t bitmask_surf, const int start)
{
	// Calculate corresponding uint in bitmask.
	const int id = blockIdx.x*blockDim.x + threadIdx.x;

	// Fill bitmask with zeros.
	if(id < count)
	{
		const unsigned int mask = 0;
		surf1Dwrite(mask, bitmask_surf, id*4);
	} 
	// Set bit corresponding to starting vertex to 1.
	if(id == (start / (8 * sizeof(unsigned int))))
	{
		const unsigned int mask = 1 << (start % (8 * sizeof(unsigned int)));
		surf1Dwrite(mask,bitmask_surf, id*4);
	}
}

void init_graph(csr::matrix graph, int*&d_row_offset, int*&d_column_index)
{
	// Allocate device memory.
	checkCudaErrors(hipMalloc((void**)&d_row_offset,(graph.n+1) * sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&d_column_index,graph.nnz * sizeof(int)));

	// Copy graph to device memory.
	checkCudaErrors(hipMemcpy(d_row_offset, graph.ptr, (graph.n+1) * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_column_index, graph.index, graph.nnz * sizeof(int), hipMemcpyHostToDevice));
}

void dispose_graph(int*& d_row_offset, int*& d_column_index)
{
	// Free device memory.
	checkCudaErrors(hipFree(d_row_offset));
	checkCudaErrors(hipFree(d_column_index));
}

void init_dist_vector(const int n, const int source_vertex, int*& d_distance)
{
	// Allocate device memory.
	checkCudaErrors(hipMalloc((void**)&d_distance,n * sizeof(int)));

	// Calculate number of blocks needed to fill distance vector.
	const int num_of_blocks = div_up(n,BLOCK_SIZE);

	// Run kernel.
	init_dist_kernel<<<num_of_blocks,BLOCK_SIZE>>>(n, d_distance,source_vertex);
}

void dispose_distance_vector(int* d_distance)
{
	// Free device memory.
	checkCudaErrors(hipFree(d_distance));
}

void init_queue_empty(const int n, int*& d_queue, int*& queue_count)
{
	// Allocate device memory.
	checkCudaErrors(hipMalloc((void**)&d_queue,n * sizeof(int)));
	checkCudaErrors(hipMallocManaged((void**)&queue_count,sizeof(int)));
	checkCudaErrors(hipDeviceSynchronize()); // needed, can get bus error without it
	// Set queue count to correct value.
	*queue_count = 0;
}
void init_queue_with_vertex(const int n, int*& d_queue, int*& queue_count, const int source_vertex)
{
	init_queue_empty(n, d_queue, queue_count);
	// Insert starting vertex into the queue.
	checkCudaErrors(hipMemcpy(d_queue, &source_vertex, sizeof(int), hipMemcpyHostToDevice));
	// Set queue count to correct value.
	*queue_count = 1;
}

void init_queue_with_edges(const int m, int*& d_queue, int*& queue_count, const int * const d_column_index, const int r, const int r_end)
{
	// Initialize empty queue.
	init_queue_empty(m, d_queue, queue_count);

	// Copy neighbors of starting vertex into the queue.
	const int count = r_end - r;
	checkCudaErrors(hipMemcpy(d_queue, d_column_index + r, count * sizeof(int), hipMemcpyDeviceToDevice));

	// Set queue count to correct value.
	*queue_count = count;
}

void dispose_queue(int*& d_queue, int*& queue_count)
{
	// Free unified memory.
	checkCudaErrors(hipFree(queue_count));

	// Free device memory.
	checkCudaErrors(hipFree(d_queue));
}

void initialize_bitmask(const int n,hipSurfaceObject_t& bitmask_surf, int source_vertex)
{
	bitmask_surf = 0;
	return;
	// problem is surface can be bound only to hipArray and with maximum width of 65536 bytes
	// make it 2d or sth idk
	/*
	   const int count = div_up(n, 8*sizeof(unsigned int));	
	   hipResourceDesc res_desc;
	   std::fill_n((volatile char*)&res_desc,sizeof(res_desc),0);

	   hipChannelFormatDesc channel_desc = hipCreateChannelDesc<unsigned int>();
	   hipArray
	   hipArray *bitmask_array;
	   checkCudaErrors(hipMallocArray(&bitmask_array, &channel_desc,count,0,hipArraySurfaceLoadStore));
	   res_desc.resType = hipResourceTypeArray;
	   res_desc.res.array.array= bitmask_array;

	   checkCudaErrors(hipCreateSurfaceObject(&bitmask_surf, &res_desc));
	   init_bitmask<<<div_up(count,BLOCK_SIZE),BLOCK_SIZE>>>(count, bitmask_surf,source_vertex);
	 */
}

void dispose_bitmask(hipSurfaceObject_t bitmask_surf)
{
	/*
	   hipResourceDesc res_desc;
	   checkCudaErrors(cudaGetSurfaceObjectResourceDesc(&res_desc, bitmask_surf));
	   checkCudaErrors(hipFreeArray(res_desc.res.array.array));
	   checkCudaErrors(hipDestroySurfaceObject(bitmask_surf));
	 */
}


bfs::result run_linear_bfs(const csr::matrix graph, int source_vertex)
{
	// Initialize graph in device memory.
	int *d_row_offset, *d_column_index;
	init_graph(graph,d_row_offset,d_column_index);

	// Initialize distance vector in device memory.
	int *d_distance;
	init_dist_vector(graph.n, source_vertex, d_distance);

	// Initialize in queue and out queue.
	int *in_queue_count, *out_queue_count;
	int *d_in_queue, *d_out_queue;
	init_queue_with_vertex(graph.n, d_in_queue, in_queue_count, source_vertex);
	init_queue_empty(graph.n, d_out_queue, out_queue_count);

	// Create events for time measurement.
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Start time measurement.
	hipEventRecord(start);

	// Start profiling.
	hipProfilerStart();
	// Algorithm
	int iteration = 0;
	while(*in_queue_count > 0)
	{
		// Empty out queue.
		*out_queue_count = 0;

		// Calculate number of blocks needed so every vertex in queue gets one thread.
		const int num_of_blocks = div_up(*in_queue_count,BLOCK_SIZE);

		// Run kernel.
		linear_bfs<<<num_of_blocks,BLOCK_SIZE>>>(graph.n,d_row_offset,d_column_index,d_distance,iteration, d_in_queue,in_queue_count,d_out_queue, out_queue_count);
		checkCudaErrors(hipDeviceSynchronize());

		iteration++;
		// Swap in and out queue. 
		std::swap(d_in_queue,d_out_queue);
		std::swap(in_queue_count, out_queue_count);
	}
	// Stop profiling.
	hipProfilerStop();

	// Calculate elapsed time.
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float miliseconds = 0;
	hipEventElapsedTime(&miliseconds, start, stop);

	// Cleanup events. 
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Copy distance vector to host memory.
	int *h_distance = new int[graph.n];
	checkCudaErrors(hipMemcpy(h_distance,d_distance,graph.n*sizeof(int),hipMemcpyDeviceToHost));

	// Free queue memory.
	dispose_queue(d_in_queue, in_queue_count);
	dispose_queue(d_out_queue, out_queue_count);
	// Free distance vector memory.
	dispose_distance_vector(d_distance); 
	// Free graph memory.
	dispose_graph(d_row_offset, d_column_index);

	// Fill and return result struct.
	bfs::result result;
	result.distance= h_distance;
	result.total_time = miliseconds;
	return result;
}

bfs::result run_quadratic_bfs(const csr::matrix graph, int source_vertex)
{
	// Initialize graph in device memory.
	int *d_row_offset, *d_column_index;
	init_graph(graph,d_row_offset,d_column_index);

	// Initialize distance vector in device memory.
	int *d_distance;
	init_dist_vector(graph.n, source_vertex, d_distance);

	// Allocate and map bool flag, used in the algorithm
	bool *done;
	checkCudaErrors(hipMallocManaged((void**)&done,sizeof(bool)));
	checkCudaErrors(hipDeviceSynchronize());
	int iteration = 0;

	// Create events for time measurement.
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Start time measurement.
	hipEventRecord(start);

	// Algorithm
	// Calculate number of blocks needed so every vertex in graph gets one thread.
	const int num_of_blocks = div_up(graph.n, BLOCK_SIZE);
	do
	{
		*done=true;
		// Run kernel.
		quadratic_bfs<<<num_of_blocks,BLOCK_SIZE>>>(graph.n,d_row_offset,d_column_index,d_distance,iteration, done);
		checkCudaErrors(hipDeviceSynchronize());
		iteration++;
	} while(!(*done));

	// Calculate elapsed time.
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float miliseconds = 0;
	hipEventElapsedTime(&miliseconds, start, stop);

	// Cleanup events. 
	hipEventDestroy(start);
	hipEventDestroy(stop);


	// Copy distance vector to host memory.
	int *h_distance = new int[graph.n];
	checkCudaErrors(hipMemcpy(h_distance,d_distance,graph.n*sizeof(int),hipMemcpyDeviceToHost));

	// Free flag memory.
	checkCudaErrors(hipFree(done));
	// Free distance vector memory.
	dispose_distance_vector(d_distance); 
	// Free graph memory.
	dispose_graph(d_row_offset, d_column_index);

	// Fill and return result struct.
	bfs::result result;
	result.distance= h_distance;
	result.total_time = miliseconds;
	return result;
}

bfs::result run_expand_contract_bfs(csr::matrix graph, int source_vertex)
{
	// Initialize graph in device memory.
	int *d_row_offset, *d_column_index;
	init_graph(graph,d_row_offset,d_column_index);

	// Initialize distance vector in device memory.
	int *d_distance;
	init_dist_vector(graph.n, source_vertex, d_distance);

	// Initialize in queue and out queue.
	int *in_queue_count, *out_queue_count;
	int *d_in_queue, *d_out_queue;
	init_queue_with_vertex(graph.n, d_in_queue, in_queue_count, source_vertex);
	init_queue_empty(graph.n, d_out_queue, out_queue_count);

	// Allocate and initialize status lookup bitmask. 
	hipSurfaceObject_t bitmask_surf = 0;
	initialize_bitmask(graph.n,bitmask_surf,source_vertex);

	// Create events for time measurement.
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Start time measurement.
	hipEventRecord(start);
	hipEventSynchronize(start);
	
	// Start profiling.
	hipProfilerStart();

	// Algorithm
	int iteration = 0;
	while(*in_queue_count > 0)
	{
		// Empty out queue.
		*out_queue_count = 0;

		// Calculate number of blocks needed so very vertex in queue gets one thread.
		const int num_of_blocks = div_up(*in_queue_count,BLOCK_SIZE);

		// Run kernel
		expand_contract_bfs<<<num_of_blocks,BLOCK_SIZE>>>(graph.n,d_row_offset,d_column_index,d_distance,iteration, d_in_queue,in_queue_count, d_out_queue, out_queue_count,bitmask_surf);
		checkCudaErrors(hipDeviceSynchronize());

		iteration++;
		// Swap in and out queue. 
		std::swap(d_in_queue,d_out_queue);
		std::swap(in_queue_count,out_queue_count);

	}

	// Stop profiling.
	hipProfilerStop();

	// Calculate elapsed time.
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float miliseconds = 0;
	hipEventElapsedTime(&miliseconds, start, stop);

	// Cleanup events. 
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Copy distance vector to host memory.
	int *h_distance = new int[graph.n];
	checkCudaErrors(hipMemcpy(h_distance,d_distance,graph.n*sizeof(int),hipMemcpyDeviceToHost));

	// Free bitmask.
	dispose_bitmask(bitmask_surf);
	// Free queue memory.
	dispose_queue(d_in_queue, in_queue_count);
	dispose_queue(d_out_queue, out_queue_count);
	// Free distance vector memory.
	dispose_distance_vector(d_distance); 
	// Free graph memory.
	dispose_graph(d_row_offset, d_column_index);

	// Fill and return result struct.
	bfs::result result;
	result.distance= h_distance;
	result.total_time = miliseconds;
	return result;
}

bfs::result run_contract_expand_bfs(csr::matrix graph, int source_vertex)
{
	// Initialize graph in device memory.
	int *d_row_offset, *d_column_index;
	init_graph(graph,d_row_offset,d_column_index);

	// Initialize distance vector in device memory.
	int *d_distance;
	init_dist_vector(graph.n, source_vertex, d_distance);

	// Initialize in queue and out queue.
	int *in_queue_count, *out_queue_count;
	int *d_in_queue, *d_out_queue;
	init_queue_with_edges(graph.nnz, d_in_queue, in_queue_count, d_column_index, graph.ptr[source_vertex], graph.ptr[source_vertex+1]);
	init_queue_empty(graph.nnz, d_out_queue, out_queue_count);

	// Create events for time measurement.
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Start time measurement.
        hipEventRecord(start);
	hipEventSynchronize(start);

	// Start profiling.
	hipProfilerStart();

	// Algorithm
	int iteration = 0;
	while(*in_queue_count > 0)
	{
		// Empty out queue.
		*out_queue_count = 0;
		checkCudaErrors(hipDeviceSynchronize());

		// Calculate number of blocks needed so every edge in queue gets one thread.
		const int num_of_blocks = div_up(*in_queue_count,BLOCK_SIZE);

		std::cout << "========" << num_of_blocks << "=============" << std::endl;
		//std::cout <<"in: " << *in_queue_count << std::endl;
		// Run kernel
		contract_expand_bfs<<<num_of_blocks,BLOCK_SIZE>>>(graph.nnz, d_row_offset, d_column_index, d_distance, iteration, d_in_queue, in_queue_count, d_out_queue, out_queue_count);
		checkCudaErrors(hipDeviceSynchronize());
		//std::cout << "out: " << *out_queue_count << std::endl;

		iteration++;
		// Swap in and out queue. 
		std::swap(d_in_queue,d_out_queue);
		std::swap(in_queue_count,out_queue_count);

	}

	// Stop profiling.
	hipProfilerStop();

	// Calculate elapsed time.
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float miliseconds = 0;
	hipEventElapsedTime(&miliseconds, start, stop);

	// Cleanup events.
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Copy distance vector to host memory.
	int *h_distance = new int[graph.n];
	checkCudaErrors(hipMemcpy(h_distance,d_distance,graph.n*sizeof(int),hipMemcpyDeviceToHost));

	// Free queue memory.
	dispose_queue(d_in_queue, in_queue_count);
	dispose_queue(d_out_queue, out_queue_count);
	// Free distance vector memory.
	dispose_distance_vector(d_distance); 
	// Free graph memory.
	dispose_graph(d_row_offset, d_column_index);

	// Fill result struct.
	bfs::result result;
	result.distance= h_distance;
	result.total_time = miliseconds;
	return result;
}
