#include "hip/hip_runtime.h"
#include "bfs.cuh"
#include "bfs_kernels.cuh"
#include "csr_matrix.h"

#include <algorithm>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

int div_up(int dividend, int divisor)
{
	return (dividend % divisor == 0)?(dividend/divisor):(dividend/divisor+1);
}

__global__ void init_distance(const int n, int*const distance,const int start)
{
	// Calculate corresponding vertex
	const int id = blockIdx.x*blockDim.x + threadIdx.x;

	// Fill distance vector
	if(id < n)
		distance[id]=bfs::infinity;
	if(id == start)
		distance[id]=0;
}

__global__ void init_bitmask(const int count, hipSurfaceObject_t bitmask_surf, const int start)
{
	// Calculate corresponding uint in bitmask
	const int id = blockIdx.x*blockDim.x + threadIdx.x;

	// Fill bitmask
	if(id < count)
	{
		const unsigned int mask = 0;
		surf1Dwrite(mask, bitmask_surf, id*4);
	} 
	if(id == (start / (8 * sizeof(unsigned int))))
	{
		const unsigned int mask = 1 << (start % (8 * sizeof(unsigned int)));
		surf1Dwrite(mask,bitmask_surf, id*4);
	}
}

void initialize_graph(csr::matrix graph, int*&d_row_offset, int*&d_column_index)
{
	// Allocate device memory
	checkCudaErrors(hipMalloc((void**)&d_row_offset,(graph.n+1) * sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&d_column_index,graph.nnz * sizeof(int)));

	// Copy graph to device memory
	checkCudaErrors(hipMemcpy(d_row_offset, graph.ptr, (graph.n+1) * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_column_index, graph.index, graph.nnz * sizeof(int), hipMemcpyHostToDevice));
}

void dispose_graph(int*& d_row_offset, int*& d_column_index)
{
	// Free device memory
	checkCudaErrors(hipFree(d_row_offset));
	checkCudaErrors(hipFree(d_column_index));
}

void initialize_distance_vector(const int n, const int starting_vertex, int*& d_distance)
{
	// Allocate device memory
	checkCudaErrors(hipMalloc((void**)&d_distance,n * sizeof(int)));

	// Calculate numbeer of blocks
	const int num_of_blocks = div_up(n,BLOCK_SIZE);

	// Run kernel initializng distance vector
	init_distance<<<num_of_blocks,BLOCK_SIZE>>>(n, d_distance,starting_vertex);
}

void dispose_distance_vector(int* d_distance)
{
	// Free device memory
	checkCudaErrors(hipFree(d_distance));
}

void initialize_vertex_queue(const int n, const int starting_vertex, int*& d_in_queue, int*& in_queue_count, int*& d_out_queue, int*& out_queue_count)
{
	// Allocate device memory
	checkCudaErrors(hipMalloc((void**)&d_in_queue,n * sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&d_out_queue,n * sizeof(int)));

	// Allocate counters as unified memory
	checkCudaErrors(hipMallocManaged((void**)&in_queue_count,sizeof(int)));
	checkCudaErrors(hipMallocManaged((void**)&out_queue_count,sizeof(int)));

	// Insert starting vertex into queue
	checkCudaErrors(hipMemcpy(d_in_queue, &starting_vertex, sizeof(int), hipMemcpyHostToDevice));

	checkCudaErrors(hipDeviceSynchronize()); // without this you can get bus error sometimes (try -qle kron_g500
	*in_queue_count=1;
	*out_queue_count=0;

}

void initialize_edge_queue(const int m, int*& d_queue, int*& queue_count)
{
	// Allocate device memory
	checkCudaErrors(hipMalloc((void**)&d_queue, m * sizeof(int)));

	// Allocate counters as unified memory
	checkCudaErrors(hipMallocManaged((void**)&queue_count,sizeof(int)));
	checkCudaErrors(hipDeviceSynchronize()); // without this you can get bus error sometimes (try -qle kron_g500
	*queue_count = 0;
}

void initialize_edge_queue(const int m, int*& d_queue, int*& queue_count, const int * const d_column_index, const int r, const int r_end)
{
	initialize_edge_queue(m, d_queue, queue_count);
	checkCudaErrors(hipMemcpy(d_queue, d_column_index + r, (r_end-r) * sizeof(int), hipMemcpyDeviceToDevice));
	*queue_count = r_end - r;
}

void dispose_edge_queue(int*& d_queue, int*& queue_count)
{
	// Free unified memory
	checkCudaErrors(hipFree(queue_count));
	// Free device memory
	checkCudaErrors(hipFree(d_queue));
}
void dispose_vertex_queue(int*& d_in_queue, int*& in_queue_count, int*& d_out_queue, int*& out_queue_count)
{
	// Free unified memory
	checkCudaErrors(hipFree(in_queue_count));
	checkCudaErrors(hipFree(out_queue_count));

	// Free device memory
	checkCudaErrors(hipFree(d_in_queue));
	checkCudaErrors(hipFree(d_out_queue));
}

void initialize_bitmask(const int n,hipSurfaceObject_t& bitmask_surf, int starting_vertex)
{
	bitmask_surf = 0;
	return;
	// problem is surface can be bound only to hipArray and with maximum width of 65536 bytes
	// make it 2d or sth idk
	/*
	   const int count = div_up(n, 8*sizeof(unsigned int));	
	   hipResourceDesc res_desc;
	   std::fill_n((volatile char*)&res_desc,sizeof(res_desc),0);

	   hipChannelFormatDesc channel_desc = hipCreateChannelDesc<unsigned int>();
	   hipArray
	   hipArray *bitmask_array;
	   checkCudaErrors(hipMallocArray(&bitmask_array, &channel_desc,count,0,hipArraySurfaceLoadStore));
	   res_desc.resType = hipResourceTypeArray;
	   res_desc.res.array.array= bitmask_array;

	   checkCudaErrors(hipCreateSurfaceObject(&bitmask_surf, &res_desc));
	   init_bitmask<<<div_up(count,BLOCK_SIZE),BLOCK_SIZE>>>(count, bitmask_surf,starting_vertex);
	 */
}

void dispose_bitmask(hipSurfaceObject_t bitmask_surf)
{
	/*
	   hipResourceDesc res_desc;
	   checkCudaErrors(cudaGetSurfaceObjectResourceDesc(&res_desc, bitmask_surf));
	   checkCudaErrors(hipFreeArray(res_desc.res.array.array));
	   checkCudaErrors(hipDestroySurfaceObject(bitmask_surf));
	 */
}


bfs::result run_linear_bfs(const csr::matrix graph, int starting_vertex)
{
	// Allocate device memory for graph and copy it
	int *d_row_offset, *d_column_index;
	initialize_graph(graph,d_row_offset,d_column_index);

	// Allocate and initialize distance vector
	int *d_distance;
	initialize_distance_vector(graph.n, starting_vertex, d_distance);

	// Allocate and initialize queues and queue counters
	int *in_queue_count, *out_queue_count;
	int *d_in_queue, *d_out_queue;
	initialize_vertex_queue(graph.n, starting_vertex, d_in_queue, in_queue_count,  d_out_queue, out_queue_count); 

	// Create events for time measurement
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Start time measurement
	hipEventRecord(start);
	hipProfilerStart();
	// Algorithm

	int iteration = 0;
	while(*in_queue_count > 0)
	{

		// Empty out queue
		*out_queue_count = 0;

		// Calculate number of blocks
		const int num_of_blocks = div_up(*in_queue_count,BLOCK_SIZE);

		// Run kernel
		linear_bfs<<<num_of_blocks,BLOCK_SIZE>>>(graph.n,d_row_offset,d_column_index,d_distance,iteration, d_in_queue,in_queue_count, d_out_queue, out_queue_count);
		checkCudaErrors(hipDeviceSynchronize());

		iteration++;
		// Swap queues
		std::swap(d_in_queue,d_out_queue);
		std::swap(in_queue_count,out_queue_count);

	}

	hipProfilerStop();
	// Calculate elapsed time
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float miliseconds = 0;
	hipEventElapsedTime(&miliseconds, start, stop);

	// Event cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Copy distance vector to host memory
	int *h_distance = new int[graph.n];
	checkCudaErrors(hipMemcpy(h_distance,d_distance,graph.n*sizeof(int),hipMemcpyDeviceToHost));

	// Free queue memory
	dispose_vertex_queue(d_in_queue, in_queue_count, d_out_queue, out_queue_count);
	// Free distance vector memory
	dispose_distance_vector(d_distance); 
	// Free graph memory
	dispose_graph(d_row_offset, d_column_index);

	bfs::result result;
	result.distance= h_distance;
	result.total_time = miliseconds;
	return result;
}

bfs::result run_quadratic_bfs(const csr::matrix graph, int starting_vertex)
{
	// Allocate device memory for graph and copy it
	int *d_row_offset, *d_column_index;
	initialize_graph(graph,d_row_offset,d_column_index);

	// Allocate and initialize distance vector
	int *d_distance;
	initialize_distance_vector(graph.n, starting_vertex, d_distance);

	// Allocate and map bool flag, for use in algorithm
	bool *h_done, *d_done;
	int iteration = 0;
	checkCudaErrors(hipHostAlloc((void**)&h_done,sizeof(bool),hipHostMallocMapped));
	checkCudaErrors(hipHostGetDevicePointer((void**)&d_done,(void*)h_done,0));

	// Create events for time measurement
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Start time measurement
	hipEventRecord(start);

	// Algorithm
	const int num_of_blocks = div_up(graph.n, BLOCK_SIZE);
	do
	{
		*h_done=true;
		quadratic_bfs<<<num_of_blocks,BLOCK_SIZE>>>(graph.n,d_row_offset,d_column_index,d_distance,iteration, d_done);
		checkCudaErrors(hipDeviceSynchronize());
		iteration++;
	} while(!(*h_done));

	// Calculate elapsed time
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float miliseconds = 0;
	hipEventElapsedTime(&miliseconds, start, stop);

	// Event cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);


	// Copy distance vector to host memory
	int *h_distance = new int[graph.n];
	checkCudaErrors(hipMemcpy(h_distance,d_distance,graph.n*sizeof(int),hipMemcpyDeviceToHost));

	// Free flag memory
	checkCudaErrors(hipHostFree(h_done));
	// Free distance vector memory
	dispose_distance_vector(d_distance); 
	// Free graph memory
	dispose_graph(d_row_offset, d_column_index);

	bfs::result result;
	result.distance= h_distance;
	result.total_time = miliseconds;
	return result;
}

bfs::result run_expand_contract_bfs(csr::matrix graph, int starting_vertex)
{
	// Allocate device memory for graph and copy it
	int *d_row_offset, *d_column_index;
	initialize_graph(graph,d_row_offset,d_column_index);

	// Allocate and initialize distance vector
	int *d_distance;
	initialize_distance_vector(graph.n, starting_vertex, d_distance);

	// Allocate and initialize queues and queue counters
	int *in_queue_count, *out_queue_count;
	int *d_in_queue, *d_out_queue;
	initialize_vertex_queue(graph.n, starting_vertex, d_in_queue, in_queue_count,  d_out_queue, out_queue_count); 

	// Allocate and initialize bitmask for status lookup
	hipSurfaceObject_t bitmask_surf = 0;
	initialize_bitmask(graph.n,bitmask_surf,starting_vertex);


	// Create events for time measurement
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Start time measurement
	hipEventRecord(start);
	hipEventSynchronize(start);
	hipProfilerStart();
	// Algorithm

	int iteration = 0;
	while(*in_queue_count > 0)
	{
		// Empty out queue
		*out_queue_count = 0;

		// Calculate number of blocks
		const int num_of_blocks = div_up(*in_queue_count,BLOCK_SIZE);

		// Run kernel
		expand_contract_bfs<<<num_of_blocks,BLOCK_SIZE>>>(graph.n,d_row_offset,d_column_index,d_distance,iteration, d_in_queue,in_queue_count, d_out_queue, out_queue_count,bitmask_surf);
		checkCudaErrors(hipDeviceSynchronize());

		iteration++;
		// Swap queues
		std::swap(d_in_queue,d_out_queue);
		std::swap(in_queue_count,out_queue_count);

	}

	hipProfilerStop();
	// Calculate elapsed time
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float miliseconds = 0;
	hipEventElapsedTime(&miliseconds, start, stop);

	// Event cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Copy distance vector to host memory
	int *h_distance = new int[graph.n];
	checkCudaErrors(hipMemcpy(h_distance,d_distance,graph.n*sizeof(int),hipMemcpyDeviceToHost));

	// Free bitmask
	dispose_bitmask(bitmask_surf);
	// Free queue memory
	dispose_vertex_queue(d_in_queue, in_queue_count, d_out_queue, out_queue_count);
	// Free distance vector memory
	dispose_distance_vector(d_distance); 
	// Free graph memory
	dispose_graph(d_row_offset, d_column_index);

	bfs::result result;
	result.distance= h_distance;
	result.total_time = miliseconds;
	return result;
}

bfs::result run_contract_expand_bfs(csr::matrix graph, int starting_vertex)
{
	// Allocate device memory for graph and copy it
	int *d_row_offset, *d_column_index;
	initialize_graph(graph,d_row_offset,d_column_index);

	// Allocate and initialize distance vector
	int *d_distance;
	initialize_distance_vector(graph.n, starting_vertex, d_distance);

	// Allocate and initialize queues and queue counters
	int *in_queue_count, *out_queue_count;
	int *d_in_queue, *d_out_queue;
	initialize_edge_queue(graph.nnz, d_in_queue, in_queue_count, d_column_index, graph.ptr[starting_vertex], graph.ptr[starting_vertex+1]);
	initialize_edge_queue(graph.nnz, d_out_queue, out_queue_count);

	// Create events for time measurement
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Start time measurement
    hipEventRecord(start);
	hipEventSynchronize(start);
	hipProfilerStart();
	checkCudaErrors(hipDeviceSynchronize());
	// Algorithm

	int iteration = 0;
	while(*in_queue_count > 0)
	{
		// Empty out queue
		*out_queue_count = 0;

		//checkCudaErrors(hipMemPrefetchAsync((void*)out_queue_count,sizeof(int),device));
		//checkCudaErrors(hipMemPrefetchAsync((void*)in_queue_count,sizeof(int),device));
		// Calculate number of blocks
		const int num_of_blocks = div_up(*in_queue_count,BLOCK_SIZE);

		//std::cout << "=====================" << std::endl;
		//std::cout <<"in: " << *in_queue_count << std::endl;
		// Run kernel
		contract_expand_bfs<<<num_of_blocks,BLOCK_SIZE>>>(graph.n, d_row_offset, d_column_index, d_distance, iteration, d_in_queue, in_queue_count, d_out_queue, out_queue_count);
		checkCudaErrors(hipDeviceSynchronize());
		//std::cout << "out: " << *out_queue_count << std::endl;

		iteration++;
		// Swap queues
		std::swap(d_in_queue,d_out_queue);
		std::swap(in_queue_count,out_queue_count);

	}
	hipProfilerStop();

	// Calculate elapsed time
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float miliseconds = 0;
	hipEventElapsedTime(&miliseconds, start, stop);

	// Event cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Copy distance vector to host memory
	int *h_distance = new int[graph.n];
	checkCudaErrors(hipMemcpy(h_distance,d_distance,graph.n*sizeof(int),hipMemcpyDeviceToHost));

	// Free queue memory
	dispose_edge_queue(d_in_queue, in_queue_count);
	dispose_edge_queue(d_out_queue, out_queue_count);
	// Free distance vector memory
	dispose_distance_vector(d_distance); 
	// Free graph memory
	dispose_graph(d_row_offset, d_column_index);

	// Fill result struct
	bfs::result result;
	result.distance= h_distance;
	result.total_time = miliseconds;
	return result;
}

