#include "hip/hip_runtime.h"
#include "common.h"
#include "bfs.cuh"
#include "csr_matrix.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

constexpr size_t WARP_SIZE= 32;
constexpr size_t BLOCK_SIZE = 512;
constexpr size_t WARPS = BLOCK_SIZE / WARP_SIZE;
constexpr size_t HASH_RANGE = 128;


// Calculate number of needed blocks
int div_up(int dividend, int divisor)
{
	return (dividend % divisor == 0)?(dividend/divisor):(dividend/divisor+1);
}

__global__ void init_distance(const int n, int*const distance,const int start)
{
	// Calculate corresponding vertex
	int id = blockIdx.x*blockDim.x + threadIdx.x;

	// Fill distance vector
	if(id < n)
		distance[id]=bfs::infinity;
	if(id == start)
		distance[id]=0;
}

void initialize_graph(csr::matrix graph, int*&d_row_offset, int*&d_column_index)
{
	// Allocate device memory
	checkCudaErrors(hipMalloc((void**)&d_row_offset,(graph.n+1) * sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&d_column_index,graph.nnz * sizeof(int)));

	// Copy graph to device memory
	checkCudaErrors(hipMemcpy(d_row_offset, graph.ptr, (graph.n+1) * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_column_index, graph.index, graph.nnz * sizeof(int), hipMemcpyHostToDevice));
}

void dispose_graph(int*& d_row_offset, int*& d_column_index)
{
	// Free device memory
	checkCudaErrors(hipFree(d_row_offset));
	checkCudaErrors(hipFree(d_column_index));
}

void initialize_distance_vector(const int n, const int starting_vertex, int*& d_distance)
{
	// Allocate device memory
	checkCudaErrors(hipMalloc((void**)&d_distance,n * sizeof(int)));

	// Calculate numbeer of blocks
	int num_of_blocks = div_up(n,BLOCK_SIZE);

	// Run kernel initializng distance vector
	init_distance<<<num_of_blocks,BLOCK_SIZE>>>(n, d_distance,starting_vertex);
}

void dispose_distance_vector(int* d_distance)
{
	// Free device memory
	checkCudaErrors(hipFree(d_distance));
}

void initialize_vertex_queue(const int n, const int starting_vertex, int*& d_in_queue, int*& d_in_queue_count, int*& h_in_queue_count, int*& d_out_queue, int*& d_out_queue_count, int*& h_out_queue_count)
{
	// Allocate device memory
	checkCudaErrors(hipMalloc((void**)&d_in_queue,n * sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&d_out_queue,n * sizeof(int)));

	// Allocate host memory and map it to device memory
	checkCudaErrors(hipHostAlloc((void**)&h_in_queue_count,sizeof(int),hipHostMallocMapped));
	checkCudaErrors(hipHostGetDevicePointer((void**)&d_in_queue_count,(void*)h_in_queue_count,0));
	checkCudaErrors(hipHostAlloc((void**)&h_out_queue_count,sizeof(int),hipHostMallocMapped));
	checkCudaErrors(hipHostGetDevicePointer((void**)&d_out_queue_count,(void*)h_out_queue_count,0));

	// Insert starting vertex into queue
	checkCudaErrors(hipMemcpy(d_in_queue, &starting_vertex, sizeof(int), hipMemcpyHostToDevice));
	*h_in_queue_count=1;
	*h_out_queue_count=0;

}

void dispose_vertex_queue(int*& d_in_queue, int*& h_in_queue_count, int*& d_out_queue, int*& h_out_queue_count)
{
	// Free host memory
	checkCudaErrors(hipHostFree(h_in_queue_count));
	checkCudaErrors(hipHostFree(h_out_queue_count));

	// Free device memory
	checkCudaErrors(hipFree(d_in_queue));
	checkCudaErrors(hipFree(d_out_queue));
}

__global__ void quadratic_bfs(const int n, const int* row_offset, const int* column_index, int*const distance, const int iteration, bool*const done)
{
	// Calculate corresponding vertex
	int id = blockIdx.x*blockDim.x + threadIdx.x;

	if(id < n && distance[id] == iteration)
	{
		bool local_done=true;
		for(int offset = row_offset[id]; offset < row_offset[id+1]; offset++)
		{
			int j = column_index[offset];
			if(distance[j] > iteration+1)
			{
				distance[j]=iteration+1;
				local_done=false;
			}
		}
		if(!local_done)
			*done=local_done;
	}
}

__global__ void linear_bfs(const int n, const int* row_offset, const int*const column_index, int*const distance, const int iteration,const int*const in_queue,const int*const in_queue_count, int*const out_queue, int*const out_queue_count)
{

	// Calculate corresponding vertex in queue
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	if(id < *in_queue_count) 
    {
        // Get vertex from the queue
        int v = in_queue[id];
        for(int offset = row_offset[v]; offset < row_offset[v+1]; offset++)
        {
            int j = column_index[offset];
            if(distance[j] == bfs::infinity)
            {
                distance[j]=iteration+1;
                // Locekd enqueue
                int ind = atomicAdd(out_queue_count,1);
                out_queue[ind]=j;
            }
        }
    }

}

__device__ bool warp_cull(volatile int scratch[WARPS][HASH_RANGE], const int v)
{
	const int hash = v & (HASH_RANGE-1);
	const int warp_id = threadIdx.x / WARP_SIZE;
    if (v != -1)
    scratch[warp_id][hash] = v;
    __syncwarp();
    const int retrieved = scratch[warp_id][hash];
    if (retrieved == v)
    {
        scratch[warp_id][hash] = threadIdx.x;
    }
    __syncwarp();
    if (retrieved == v && scratch[warp_id][hash] != threadIdx.x)
    {
        return true;
    }
	return false;
}

__device__ bool history_cull()
{

	return false;
}

__device__ int2 block_prefix_sum(const int val)
{
    // Heavily inspired/copied from sample "shfl_scan" provied by NVIDIA
    // Block-wide prefix sum using shfl intrinsic
    __shared__ int sums[WARPS];
    int value = val;

    const int lane_id = threadIdx.x % WARP_SIZE;
    const int warp_id = threadIdx.x / WARP_SIZE;
    
    // Warp-wide prefix sums
#pragma unroll
    for(int i = 1; i <= WARP_SIZE; i <<= 1)
    {
        const unsigned int mask = 0xffffffff;
        const int n = __shfl_up_sync(mask, value, i, WARP_SIZE);
        if (lane_id >= i)
            value += n;
    }

    // Write warp total to shared array
    if (threadIdx.x % WARP_SIZE == WARP_SIZE- 1)
    {
        sums[warp_id] = value;
    }

    __syncthreads();

    // Prefix sum of warp sums
    if (warp_id == 0 && lane_id < (blockDim.x / WARP_SIZE))
    {
        int warp_sum = sums[lane_id];
        const unsigned int mask = (1 << (WARPS)) - 1;
#pragma unroll
        for (int i = 1; i <= WARPS; i <<= 1)
        {
            const int n = __shfl_up_sync(mask, warp_sum, i, WARPS);
            if (lane_id >= i)
                warp_sum += n;
        }

        sums[lane_id] = warp_sum;
    }

    __syncthreads();


    // Add total sum of previous warps to current element
    if (warp_id > 0)
    {
       const int block_sum = sums[warp_id-1];
       value += block_sum;
    }
    
    int2 result;
    // Subtract value given by thread to get exclusive prefix sum
    result.x = value - val;
    // Get total sum
    result.y = sums[WARPS-1];
    return result; 
}

__device__ bool status_lookup(int * const distance, int neighbor)
{
    bool is_valid = false;
     // TODO bitmask
    is_valid = distance[neighbor] == bfs::infinity;
    return is_valid;
}

__device__ void block_coarse_grained_gather(const int* const column_index, int* const distance, const int iteration, int * const out_queue, int* const out_queue_count,int r, int r_end)
{
    volatile __shared__ int comm[3];
    const int thread_id = threadIdx.x;
    //while(block_any(r_end-r))
    while(__syncthreads_or(r_end-r))
    {
        if(r_end-r)
            comm[0] = thread_id;
        __syncthreads();
        if(comm[0] == thread_id)
        {
            comm[1] = r;
            comm[2] = r_end;
            r = r_end;
        }
        __syncthreads();
        int r_gather = comm[1] + thread_id;
        const int r_gather_end = comm[2];
        int block_progress = 0;
        const int total = comm[2] - comm[1];
        while((total - block_progress) > 0)
        {
            int neighbor = -1;
            bool is_valid = false;
            if (r_gather < r_gather_end)
            {
                neighbor = column_index[r_gather];
                // Look up status
                is_valid = status_lookup(distance, neighbor);
                if(is_valid)
                {
                    // Update label
                    distance[neighbor] = iteration + 1;
                }
            }
            // Prefix sum
            const int2 queue_offset = block_prefix_sum(is_valid?1:0);
            volatile __shared__ int base_offset[1];
            // Obtain base enqueue offset
            if(threadIdx.x == 0)
                base_offset[0] = atomicAdd(out_queue_count,queue_offset.y);
            __syncthreads();
            // Write to queue
            if (is_valid)
                out_queue[base_offset[0]+queue_offset.x] = neighbor;


            r_gather += BLOCK_SIZE;
            block_progress+= BLOCK_SIZE;
            __syncthreads();
        }
    }
}

__device__ void fine_grained_gather(const int* const column_index, int* const distance, const int iteration, int * const out_queue, int* const out_queue_count,int r, int r_end)
{
    // Fine-grained neigbor-gathering
    // Prefix scan
    int2 ranks = block_prefix_sum(r_end-r);

    int rsv_rank = ranks.x;
    const int total = ranks.y;

    __shared__ int comm[BLOCK_SIZE];
    int cta_progress = 0;
    int remain;

    while ((remain = total - cta_progress) > 0)
    {
        while((rsv_rank < cta_progress + BLOCK_SIZE) && (r < r_end))
        {
            comm[rsv_rank - cta_progress] = r;
            rsv_rank++;
            r++;
        }
        __syncthreads();
        int neighbor;
        bool is_valid = false;
        if (threadIdx.x < remain && threadIdx.x < BLOCK_SIZE)
        {
            neighbor = column_index[comm[threadIdx.x]];
            // Look up status
            is_valid = status_lookup(distance, neighbor);
            if(is_valid)
            {
                // Update label
                distance[neighbor] = iteration + 1;
            }
        }
        // Prefix sum
        const int2 queue_offset = block_prefix_sum(is_valid?1:0);
        volatile __shared__ int base_offset[1];
        // Obtain base enqueue offset
        if(threadIdx.x == 0)
            base_offset[0] = atomicAdd(out_queue_count,queue_offset.y);
        __syncthreads();
        // Write to queue
        if (is_valid)
            out_queue[base_offset[0]+queue_offset.x] = neighbor;

        cta_progress += BLOCK_SIZE;
        __syncthreads();
    }
}

__global__ void expand_contract_bfs(const int n, const int* row_offset, const int* column_index, int* distance, const int iteration,const int* in_queue,const int* in_queue_count, int* out_queue, int* out_queue_count)
{
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    //if(tid >= *in_queue_count) return; // you can't do this

    int queue_count = *in_queue_count;

    // Get vertex from the queue
    const int v = tid < queue_count? in_queue[tid]:-1;

    // Local warp-culling
    volatile __shared__ int scratch[WARPS][HASH_RANGE];
    bool is_duplicate =  warp_cull(scratch, v);

    
    // Local history-culling
    // TODO
    //volatile __shared__ int history[BLOCK_SIZE][2];

    // Load corresponding row-ranges
    int r = is_duplicate?0:row_offset[v];
    int r_end = is_duplicate?0:row_offset[v+1];
    int count = r_end - r;

    // TODO Coarse-grained neighbor-gathering

    int end = count >= BLOCK_SIZE ? r_end: r;
    __syncthreads();
   // block_coarse_grained_gather(column_index, distance, iteration, out_queue, out_queue_count, r, r_end);
//    fine_grained_gather(column_index, distance, iteration, out_queue, out_queue_count, r, r_end);
    block_coarse_grained_gather(column_index, distance, iteration, out_queue, out_queue_count, r, end);
    __syncthreads();
    end = count < BLOCK_SIZE ? r_end: r;
    fine_grained_gather(column_index, distance, iteration, out_queue, out_queue_count, r,end);

}

bfs::result run_linear_bfs(const csr::matrix graph, int starting_vertex)
{
	// Allocate device memory for graph and copy it
	int *d_row_offset, *d_column_index;
	initialize_graph(graph,d_row_offset,d_column_index);

	// Allocate and initialize distance vector
	int *d_distance;
	initialize_distance_vector(graph.n, starting_vertex, d_distance);

	// Allocate and initialize queues and queue counters
	int *d_in_queue_count, *d_out_queue_count;
	int *h_in_queue_count, *h_out_queue_count;
	int *d_in_queue, *d_out_queue;
	initialize_vertex_queue(graph.n, starting_vertex, d_in_queue, d_in_queue_count, h_in_queue_count, d_out_queue, d_out_queue_count, h_out_queue_count); 

	// Create events for time measurement
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Start time measurement
	hipEventRecord(start);
	hipProfilerStart();
	// Algorithm
	*h_in_queue_count=1;
	*h_out_queue_count=0;


	int iteration = 0;
	while(*h_in_queue_count > 0)
	{

		// Empty out queue
		*h_out_queue_count = 0;

		// Calculate number of blocks
		int num_of_blocks = div_up(*h_in_queue_count,BLOCK_SIZE);

		// Run kernel
		linear_bfs<<<num_of_blocks,BLOCK_SIZE>>>(graph.n,d_row_offset,d_column_index,d_distance,iteration, d_in_queue,d_in_queue_count, d_out_queue, d_out_queue_count);
		checkCudaErrors(hipDeviceSynchronize());

		// Increment iteration counf
		iteration++;
		// Swap queues
		std::swap(d_in_queue,d_out_queue);
		std::swap(h_in_queue_count,h_out_queue_count);
		std::swap(d_in_queue_count,d_out_queue_count);

	}

	hipProfilerStop();
	// Calculate elapsed time
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float miliseconds = 0;
	hipEventElapsedTime(&miliseconds, start, stop);

	// Event cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Copy distance vector to host memory
	int *h_distance = new int[graph.n];
	checkCudaErrors(hipMemcpy(h_distance,d_distance,graph.n*sizeof(int),hipMemcpyDeviceToHost));

	// Free queue memory
	dispose_vertex_queue(d_in_queue, h_in_queue_count, d_out_queue, h_out_queue_count);
	// Free distance vector memory
	dispose_distance_vector(d_distance); 
	// Free graph memory
	dispose_graph(d_row_offset, d_column_index);

	bfs::result result;
	result.distance= h_distance;
	result.total_time = miliseconds;
	return result;
}

bfs::result run_quadratic_bfs(const csr::matrix graph, int starting_vertex)
{
	// Allocate device memory for graph and copy it
	int *d_row_offset, *d_column_index;
	initialize_graph(graph,d_row_offset,d_column_index);

	// Allocate and initialize distance vector
	int *d_distance;
	initialize_distance_vector(graph.n, starting_vertex, d_distance);

	// Allocate and map bool flag, for use in algorithm
	bool *h_done, *d_done;
	int iteration = 0;
	checkCudaErrors(hipHostAlloc((void**)&h_done,sizeof(bool),hipHostMallocMapped));
	checkCudaErrors(hipHostGetDevicePointer((void**)&d_done,(void*)h_done,0));

	// Create events for time measurement
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Start time measurement
	hipEventRecord(start);

	// Algorithm
	int num_of_blocks = div_up(graph.n, BLOCK_SIZE);
	do
	{
		*h_done=true;
		quadratic_bfs<<<num_of_blocks,BLOCK_SIZE>>>(graph.n,d_row_offset,d_column_index,d_distance,iteration, d_done);
		checkCudaErrors(hipDeviceSynchronize());
		iteration++;
	} while(!(*h_done));

	// Calculate elapsed time
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float miliseconds = 0;
	hipEventElapsedTime(&miliseconds, start, stop);

	// Event cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);


	// Copy distance vector to host memory
	int *h_distance = new int[graph.n];
	checkCudaErrors(hipMemcpy(h_distance,d_distance,graph.n*sizeof(int),hipMemcpyDeviceToHost));

	// Free flag memory
	checkCudaErrors(hipHostFree(h_done));
	// Free distance vector memory
	dispose_distance_vector(d_distance); 
	// Free graph memory
	dispose_graph(d_row_offset, d_column_index);

	bfs::result result;
	result.distance= h_distance;
	result.total_time = miliseconds;
	return result;
}

bfs::result run_expand_contract_bfs(csr::matrix graph, int starting_vertex)
{
	// Allocate device memory for graph and copy it
	int *d_row_offset, *d_column_index;
	initialize_graph(graph,d_row_offset,d_column_index);

	// Allocate and initialize distance vector
	int *d_distance;
	initialize_distance_vector(graph.n, starting_vertex, d_distance);

	// Allocate and initialize queues and queue counters
	int *d_in_queue_count, *d_out_queue_count;
	int *h_in_queue_count, *h_out_queue_count;
	int *d_in_queue, *d_out_queue;
	initialize_vertex_queue(graph.n, starting_vertex, d_in_queue, d_in_queue_count, h_in_queue_count, d_out_queue, d_out_queue_count, h_out_queue_count); 

	// Create events for time measurement
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Start time measurement
	hipEventRecord(start);
	hipEventSynchronize(start);
	hipProfilerStart();
	// Algorithm
	*h_in_queue_count=1;
	*h_out_queue_count=0;


	int iteration = 0;
	while(*h_in_queue_count > 0)
	{

		// Empty out queue
		*h_out_queue_count = 0;

		// Calculate number of blocks
		int num_of_blocks = div_up(*h_in_queue_count,BLOCK_SIZE);

		// Run kernel
		expand_contract_bfs<<<num_of_blocks,BLOCK_SIZE>>>(graph.n,d_row_offset,d_column_index,d_distance,iteration, d_in_queue,d_in_queue_count, d_out_queue, d_out_queue_count);
		checkCudaErrors(hipDeviceSynchronize());

		// Increment iteration counf
		iteration++;
		// Swap queues
		std::swap(d_in_queue,d_out_queue);
		std::swap(h_in_queue_count,h_out_queue_count);
		std::swap(d_in_queue_count,d_out_queue_count);

	}

	hipProfilerStop();
	// Calculate elapsed time
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float miliseconds = 0;
	hipEventElapsedTime(&miliseconds, start, stop);

	// Event cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Copy distance vector to host memory
	int *h_distance = new int[graph.n];
	checkCudaErrors(hipMemcpy(h_distance,d_distance,graph.n*sizeof(int),hipMemcpyDeviceToHost));

	// Free queue memory
	dispose_vertex_queue(d_in_queue, h_in_queue_count, d_out_queue, h_out_queue_count);
	// Free distance vector memory
	dispose_distance_vector(d_distance); 
	// Free graph memory
	dispose_graph(d_row_offset, d_column_index);

	bfs::result result;
	result.distance= h_distance;
	result.total_time = miliseconds;
	return result;
}
