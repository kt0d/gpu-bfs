#include "hip/hip_runtime.h"
#include "bfs_kernels.cuh"

#include "common.h"

#include <stdio.h>

constexpr unsigned int FULL_MASK = 0xffffffff;


__global__ void quadratic_bfs(const int n, const int* row_offset, const int* column_index, int*const distance, const int iteration, bool*const done)
{
	// Calculate corresponding vertex.
	const int global_tid = blockIdx.x*blockDim.x + threadIdx.x;

	// Don't go out of bounds.
	if(global_tid >= n) return;
	// Inspect only vertices in current frontier.
	if(distance[global_tid] != iteration) return;

	bool local_done=true;
	for(int offset = row_offset[global_tid]; offset < row_offset[global_tid+1]; offset++)
	{
		const int j = column_index[offset];
		if(distance[j] > iteration+1)
		{
			distance[j]=iteration+1;
			local_done=false;
		}
	}
	if(!local_done)
		*done=local_done;
}

__global__ void linear_bfs(const int n, const int* row_offset, const int*const column_index, int*const distance, const int iteration,const int*const in_queue,const int*const in_queue_count, int*const out_queue, int*const out_queue_count)
{
	// Calculate index of corresponding vertex in the queue.
	const int global_tid = blockIdx.x*blockDim.x + threadIdx.x;
	// Don't go out of bounds.
	if(global_tid >= *in_queue_count) return;
	// Get vertex from the queue.
	const int v = in_queue[global_tid];
	for(int offset = row_offset[v]; offset < row_offset[v+1]; offset++)
	{
		const int j = column_index[offset];
		if(distance[j] == bfs::infinity)
		{
			distance[j]=iteration+1;
			// Enqueue vertex.
			const int ind = atomicAdd(out_queue_count,1);
			out_queue[ind]=j;
		}
	}
}

__device__ bool warp_cull(volatile int scratch[WARPS][HASH_RANGE], const int v)
{
	const int hash = v & (HASH_RANGE-1);
	const int warp_id = threadIdx.x / WARP_SIZE;

	// Threads without valid vertex provide -1 as v. They must enter this function, because they are needed for __syncwarp. Alternative: calculate mask of threads with correct vertices and use it in __syncwarp.
	if (v != -1)
		scratch[warp_id][hash]= v;
	__syncwarp();
	const int retrieved = scratch[warp_id][hash];
	if (retrieved == v)
	{
		// Vie to be the only thread in warp inspecting vertex v.
		scratch[warp_id][hash] = threadIdx.x;
	}
	__syncwarp();
	if(v == -1)
		return true;
	if (retrieved == v && scratch[warp_id][hash] != threadIdx.x)
	{
		// Some other thread has this vertex.
		return true;
	}
	return false;
}

__device__ bool history_cull()
{
	//TODO
	return false;
}

__device__ int2 block_prefix_sum(const int val)
{
	// Heavily inspired/copied from sample "shfl_scan" provied by NVIDIA
	// Block-wide prefix sum using shfl intrinsic.
	volatile __shared__ int sums[WARPS];
	int value = val;

	const int lane_id = threadIdx.x % WARP_SIZE;
	const int warp_id = threadIdx.x / WARP_SIZE;

	// Warp-wide prefix sums.
#pragma unroll
	for(int i = 1; i <= WARP_SIZE; i <<= 1)
	{
		const int n = __shfl_up_sync(FULL_MASK, value, i, WARP_SIZE);
		if (lane_id >= i)
			value += n;
	}

	// Write warp total to shared array.
	if (threadIdx.x % WARP_SIZE == WARP_SIZE- 1)
	{
		sums[warp_id] = value;
	}

	__syncthreads();

	// Prefix sum of warp sums.
	if (warp_id == 0 && lane_id < WARPS)
	{
		int warp_sum = sums[lane_id];
		const unsigned int mask = (1 << (WARPS)) - 1;
#pragma unroll
		for (int i = 1; i <= WARPS; i <<= 1)
		{
			const int n = __shfl_up_sync(mask, warp_sum, i, WARPS);
			if (lane_id >= i)
				warp_sum += n;
		}

		sums[lane_id] = warp_sum;
	}

	__syncthreads();

	// Add total sum of previous warps to current element.
	if (warp_id > 0)
	{
		const int block_sum = sums[warp_id-1];
		value += block_sum;
	}

	int2 result;
	// Subtract value given by thread to get exclusive prefix sum.
	result.x = value - val;
	// Get total sum.
	result.y = sums[WARPS-1];
	return result; 
}

__device__ bool status_lookup(int * const distance,const hipSurfaceObject_t bitmask_surf, const int neighbor)
{
	// Just check status directly if bitmask is unavailable.
	if (bitmask_surf == 0)
		return distance[neighbor] == bfs::infinity;
	bool not_visited = false;

	const unsigned int neighbor_mask = (1 << (neighbor % (8 * sizeof(unsigned int))));
	unsigned int mask = 0;
	const int count = neighbor / (8 * sizeof(unsigned int));
	surf1Dread(&mask, bitmask_surf, count* 4);
	if(mask & neighbor_mask )
	{
		return false;
	}

	not_visited = distance[neighbor] == bfs::infinity;

	if(not_visited)
	{
		// Update bitmask
		mask |= neighbor_mask;
		surf1Dwrite(mask,bitmask_surf,count * 4);	
	}

	return not_visited;
}

__device__ void block_gather(const int* const column_index, int* const distance, hipSurfaceObject_t bitmask_surf, const int iteration, int * const out_queue, int* const out_queue_count,int r, int r_end)
{
	volatile __shared__ int comm[3];
	while(__syncthreads_or(r < r_end))
	{
		// Vie for control of block.
		if(r < r_end)
			comm[0] = threadIdx.x;
		__syncthreads();
		if(comm[0] == threadIdx.x)
		{
			// If won, share your range to the entire block.
			comm[1] = r;
			comm[2] = r_end;
			r = r_end;
		}
		__syncthreads();
		int r_gather = comm[1] + threadIdx.x;
		const int r_gather_end = comm[2];
		const int total = comm[2] - comm[1];
		int block_progress = 0;
		// TODO simplify it
		while((total - block_progress) > 0)
		{
			int neighbor = -1;
			bool is_valid = false;
			if (r_gather < r_gather_end)
			{
				neighbor = column_index[r_gather];
				// Look up status of current neighbor.
				is_valid = status_lookup(distance,bitmask_surf, neighbor);
				if(is_valid)
				{
					// Update label.
					distance[neighbor] = iteration + 1;
				}
			}
			// Obtain offset in queue by computing prefix sum
			const int2 queue_offset = block_prefix_sum(is_valid?1:0);
			volatile __shared__ int base_offset[1];

			// Obtain base enqueue offset and share it to whole block.
			if(threadIdx.x == 0)
				base_offset[0] = atomicAdd(out_queue_count,queue_offset.y);
			__syncthreads();
			// Write vertex to the out queue.
			if (is_valid)
				out_queue[base_offset[0]+queue_offset.x] = neighbor;

			r_gather += BLOCK_SIZE;
			block_progress+= BLOCK_SIZE;
			__syncthreads();
		}
	}
}

__device__ void fine_gather(const int* const column_index, int* const distance,hipSurfaceObject_t bitmask_surf, const int iteration, int * const out_queue, int* const out_queue_count,int r, int r_end)
{
	const int2 ranks = block_prefix_sum(r_end-r);

	int rsv_rank = ranks.x;
	const int total = ranks.y;

	__shared__ int comm[BLOCK_SIZE];
	int cta_progress = 0;
	int remain;

	while ((remain = total - cta_progress) > 0)
	{
		// Pack shared array with neighbors from adjacency lists.
		while((rsv_rank < cta_progress + BLOCK_SIZE) && (r < r_end))
		{
			comm[rsv_rank - cta_progress] = r;
			rsv_rank++;
			r++;
		}
		__syncthreads();
		int neighbor;
		bool is_valid = false;
		if (threadIdx.x < remain)
		{
			neighbor = column_index[comm[threadIdx.x]];
			// Look up status
			is_valid = status_lookup(distance,bitmask_surf, neighbor);
			if(is_valid)
			{
				// Update label
				distance[neighbor] = iteration + 1;
			}
		}
		__syncthreads();
		// Obtain offset in queue by computing prefix sum.
		const int2 queue_offset = block_prefix_sum(is_valid?1:0);
		volatile __shared__ int base_offset[1];
		// Obtain base enqueue offset
		if(threadIdx.x == 0)
		{
			base_offset[0] = atomicAdd(out_queue_count,queue_offset.y);
		}
		__syncthreads();
		const int queue_index = base_offset[0] + queue_offset.x;
		// Write to queue
		if (is_valid)
		{
			out_queue[queue_index] = neighbor;
		}

		cta_progress += BLOCK_SIZE;
		__syncthreads();
	}
}

__global__ void expand_contract_bfs(const int n, const int* const row_offset, const int* const column_index, int* const distance, const int iteration,const int* const in_queue,const int* const in_queue_count, int* const out_queue, int* const out_queue_count, hipSurfaceObject_t bitmask_surf)
{
	const int global_tid = blockIdx.x*blockDim.x + threadIdx.x;
	const int queue_count = *in_queue_count;

	// Get vertex from the queue.
	const int v = global_tid < queue_count? in_queue[global_tid]:-1;

	// Do local warp-culling.
	volatile __shared__ int scratch[WARPS][HASH_RANGE];
	const bool is_duplicate = warp_cull(scratch, v);

	// Load corresponding row-ranges.
	const int r = is_duplicate?0:row_offset[v];
	const int r_end = is_duplicate?0:row_offset[v+1];
	const bool big_list = (r_end - r) >= BLOCK_SIZE;

	// Both expand and contract phases occur in these functions.
	block_gather(column_index, distance,bitmask_surf, iteration, out_queue, out_queue_count, r, big_list ? r_end : r);
	fine_gather(column_index, distance,bitmask_surf, iteration, out_queue, out_queue_count, r, big_list ? r : r_end);

}

__device__ void fine_gather(const int* const column_index, int* const out_queue, int r, int r_end, int rsv_rank, const int total, const int base_offset)
{
	volatile __shared__ int comm[BLOCK_SIZE];
	int cta_progress = 0;
	int remain;
	while ((remain = total - cta_progress) > 0)
	{
		// Pack shared array with neighbors from adjacency lists.
		while((rsv_rank < cta_progress + BLOCK_SIZE) && (r < r_end))
		{
			comm[rsv_rank - cta_progress] = r;
			rsv_rank++;
			r++;
		}
		__syncthreads();
		if (threadIdx.x < remain)
		{
			const int neighbor = column_index[comm[threadIdx.x]];
			const int queue_index = base_offset+cta_progress + threadIdx.x;
			// Write to queue.
			out_queue[queue_index] = neighbor;
		}
		cta_progress += BLOCK_SIZE;
		__syncthreads();
	}
}

__device__ void warp_gather(const int* const column_index, int * const out_queue,int r, const int r_end, int rsv_rank, int base_offset)
{
	volatile __shared__ int comm[WARPS][3];
	const int lane_id = threadIdx.x % WARP_SIZE;
	const int warp_id = threadIdx.x / WARP_SIZE;
	while(__any_sync(FULL_MASK,r < r_end))
	{
		// Vie for control of warp.
		if(r < r_end)
			comm[warp_id][0] = lane_id;
		__syncwarp();
		if(comm[warp_id][0] == lane_id)
		{
			// If won, share your range and enqueue offset to the entire warp.
			__syncwarp();
			comm[warp_id][0] = rsv_rank;
			comm[warp_id][1] = r;
			comm[warp_id][2] = r_end;
			r = r_end;
		}
		__syncwarp();
		int r_gather = comm[warp_id][1] + lane_id;
		const int r_gather_end = comm[warp_id][2];
		int queue_index = base_offset+comm[warp_id][0] + lane_id;
		while(r_gather < r_gather_end)
		{
			const int v = column_index[r_gather];
			out_queue[queue_index] = v;
			r_gather += WARP_SIZE;
			queue_index += WARP_SIZE;
		}
	}
}

__global__ void contract_expand_bfs(const int n, const int* const row_offset, const int* const column_index, int* const distance, const int iteration, const int*const in_queue,const int* const in_queue_count, int* const out_queue, int* const out_queue_count)
{
	const int global_tid = blockIdx.x*blockDim.x + threadIdx.x;
	const int queue_count = *in_queue_count;

	// Get neighbor from the queue.
	const int v = global_tid < queue_count? in_queue[global_tid]:-1;

	// Contract phase: filter previously visited and duplicate neighbors.
	const bool is_valid = v != -1 ? (distance[v] == bfs::infinity) : false;
	volatile __shared__ int scratch[WARPS][HASH_RANGE];
	const bool is_duplicate = warp_cull(scratch, v);
	int r = 0, r_end = 0;
	if(is_valid && !is_duplicate)
	{
		distance[v] = iteration + 1;
		r = row_offset[v];
		r_end = row_offset[v+1];
	}

	// Expand phase: expand adjacency lists and copy them to the out queue.
	const bool big_list = (r_end - r) >= WARP_SIZE; 
	const int2 warp_gather_prescan = block_prefix_sum(big_list ? (r_end - r):0);
	__syncthreads(); // __syncthreads is very much needed because of shared array used in block_prefix_sum
	const int2 fine_gather_prescan = block_prefix_sum(big_list ? 0 : (r_end - r));
	
	volatile __shared__ int base_offset[1];
	if(threadIdx.x == 0)
		base_offset[0] = atomicAdd(out_queue_count, warp_gather_prescan.y + fine_gather_prescan.y);
	__syncthreads();
	int base = base_offset[0];	
	warp_gather(column_index, out_queue, r, big_list ? r_end : 0, warp_gather_prescan.x, base);
	base += warp_gather_prescan.y;
	fine_gather(column_index, out_queue, r, big_list ? 0: r_end, fine_gather_prescan.x, fine_gather_prescan.y, base);

}

